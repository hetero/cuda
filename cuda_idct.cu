#include "hip/hip_runtime.h"
#include <stdint.h>
#include "cuda_common.h"

__constant__ static uint8_t quanttbl[2][64] =
{
    {
        6, 4, 4, 5, 4, 4, 6, 5,
        5, 5, 7, 6, 6, 7, 9, 16, 
        10, 9, 8, 8, 9, 19, 14, 14, 
        11, 16, 23, 20, 24, 12, 22, 20, 
        22, 22, 25, 28, 36, 31, 25, 27, 
        34, 27, 22, 22, 32, 43, 32, 34, 
        38, 39, 41, 41, 41, 24, 30, 45, 
        48, 44, 40, 48, 36, 40, 41, 39
    },
    {
        6, 7, 7, 9, 8, 9, 18, 10, 
        10, 18, 39, 26, 22, 26, 39, 39, 
        39, 39, 39, 39, 39, 39, 39, 39, 
        39, 39, 39, 39, 39, 39, 39, 39, 
        39, 39, 39, 39, 39, 39, 39, 39, 
        39, 39, 39, 39, 39, 39, 39, 39, 
        39, 39, 39, 39, 39, 39, 39, 39, 
        39, 39, 39, 39, 39, 39, 39, 39
    }
};

__constant__ static uint8_t zigzag_U[64] =
{
    0,
    1, 0,
    0, 1, 2,
    3, 2, 1, 0,
    0, 1, 2, 3, 4,
    5, 4, 3, 2, 1, 0,
    0, 1, 2, 3, 4, 5, 6,
    7, 6, 5, 4, 3, 2, 1, 0,
    1, 2, 3, 4, 5, 6, 7,
    7, 6, 5, 4, 3, 2,
    3, 4, 5, 6, 7,
    7, 6, 5, 4,
    5, 6, 7,
    7, 6,
    7,
};

__constant__ static uint8_t zigzag_V[64] =
{
    0,
    0, 1,
    2, 1, 0,
    0, 1, 2, 3,
    4, 3, 2, 1, 0,
    0, 1, 2, 3, 4, 5,
    6, 5, 4, 3, 2, 1, 0,
    0, 1, 2, 3, 4, 5, 6, 7,
    7, 6, 5, 4, 3, 2, 1,
    2, 3, 4, 5, 6, 7,
    7, 6, 5, 4, 3,
    4, 5, 6, 7,
    7, 6, 5,
    6, 7,
    7,
};

__constant__ static float dctlookup[8][8] = {
    {1.000000f, 0.980785f, 0.923880f, 0.831470f, 0.707107f, 0.555570f, 0.382683f, 0.195090f, },
    {1.000000f, 0.831470f, 0.382683f, -0.195090f, -0.707107f, -0.980785f, -0.923880f, -0.555570f, },
    {1.000000f, 0.555570f, -0.382683f, -0.980785f, -0.707107f, 0.195090f, 0.923880f, 0.831470f, },
    {1.000000f, 0.195090f, -0.923880f, -0.555570f, 0.707107f, 0.831470f, -0.382683f, -0.980785f, },
    {1.000000f, -0.195090f, -0.923880f, 0.555570f, 0.707107f, -0.831470f, -0.382683f, 0.980785f, },
    {1.000000f, -0.555570f, -0.382683f, 0.980785f, -0.707107f, -0.195090f, 0.923880f, -0.831470f, },
    {1.000000f, -0.831470f, 0.382683f, 0.195090f, -0.707107f, 0.980785f, -0.923880f, 0.555570f, },
    {1.000000f, -0.980785f, 0.923880f, -0.831470f, 0.707107f, -0.555570f, 0.382683f, -0.195090f, },
};

__device__ static void cuda_scale_block(float *in_data, float *out_data, const int &col_mb)
{
#define row_mb (threadIdx.y)
    float a1 = !col_mb ? ISQRT2 : 1.0f;
    float a2 = !row_mb ? ISQRT2 : 1.0f;
    int idx = DCT_TH_X * threadIdx.y + col_mb;
    out_data[idx] = in_data[idx] * a1 * a2;
}

__device__ static void cuda_idct_1d(float *in_row, float *out_cell, const int &col_mb)
{
#define dct_col (threadIdx.y)
    float idct = in_row[0] * dctlookup[dct_col][0];
    idct += in_row[1] * dctlookup[dct_col][1];
    idct += in_row[2] * dctlookup[dct_col][2];
    idct += in_row[3] * dctlookup[dct_col][3];
    idct += in_row[4] * dctlookup[dct_col][4];
    idct += in_row[5] * dctlookup[dct_col][5];
    idct += in_row[6] * dctlookup[dct_col][6];
    idct += in_row[7] * dctlookup[dct_col][7];
    
    *out_cell = idct;
}

__device__ static void cuda_dequantize_block(float *in_data, float *out_data, uint8_t id_quant, int col_mb)
{
    int zigzag = 8 * threadIdx.y + col_mb;
    uint8_t u = zigzag_U[zigzag];
    uint8_t v = zigzag_V[zigzag];
    float dct = in_data[DCT_TH_X * threadIdx.y + col_mb];
    out_data[DCT_TH_X * v + u] = rintf((dct * quanttbl[id_quant][zigzag]) / 4.0f);
}

__device__ static void cuda_dequant_idct_block_8x8(
        float *mb, float *mb2, uint8_t id_quant,
        const int &col_mb, const int &block_pos)
{
    int first_col = (threadIdx.x >> 3) << 3;
    int first_col_row = DCT_TH_X * col_mb + first_col;
    cuda_dequantize_block(mb + first_col, mb2 + first_col, id_quant, col_mb);
    __syncthreads();
    cuda_scale_block(mb2 + first_col, mb + first_col, col_mb);
    __syncthreads();
    cuda_idct_1d(mb + first_col_row, mb2 + block_pos, col_mb);
    __syncthreads();
    cuda_idct_1d(mb2 + first_col_row, mb + block_pos, col_mb);
    __syncthreads();
}

__global__ static void k_dequant_idct_block_8x8(
        int16_t *in_data, uint8_t *prediction, uint32_t width,
        uint8_t *out_data, uint8_t id_quant)
{
    __shared__ float mb[DCT_BL_SIZE], mb2[DCT_BL_SIZE];
    int col_mb = threadIdx.x & 7;
    int nr_mb = threadIdx.x >> 3;
    int block_pos = DCT_TH_X * threadIdx.y + threadIdx.x;
    int idxIn = 8 * width * blockIdx.y + DCT_BL_SIZE * blockIdx.x + 8 * threadIdx.y + col_mb + 64 * nr_mb;
    mb[block_pos] = in_data[idxIn];
    cuda_dequant_idct_block_8x8(mb, mb2, id_quant, col_mb, block_pos);
    int idxPredOut = 8 * width * blockIdx.y + DCT_TH_X * blockIdx.x + width * threadIdx.y + threadIdx.x;
    int tmp = (int)mb[block_pos] + (int)prediction[idxPredOut];
    if (tmp < 0)
        tmp = 0;
    else if (tmp > 255)
        tmp = 255;
    out_data[idxPredOut] = tmp;
}

__host__ void cuda_dequantize_idct(uint32_t width, uint32_t height,
        uint8_t id_quant, int16_t *d_in_data, uint8_t *d_prediction,
        uint8_t *d_out_data)
{
    dim3 threadsPerBlock(DCT_TH_X, DCT_TH_Y);
    dim3 blocksPerGrid((width + DCT_TH_X - 1) / DCT_TH_X, height / 8);
    k_dequant_idct_block_8x8<<<blocksPerGrid, threadsPerBlock>>>(
            d_in_data, d_prediction, width, d_out_data, id_quant);
}

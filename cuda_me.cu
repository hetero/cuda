#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <getopt.h>
#include <errno.h>
#include <stdint.h>
#include <math.h>
#include <assert.h>
#include <limits.h>
#include <sm_11_atomic_functions.h>
#include <sm_12_atomic_functions.h>
#include <sm_13_double_functions.h>

#include "c63.h"
#include "cuda_me.h"
//#include "cuPrintf.cu"

#define REF_WIDTH 48
#define REF_HEIGHT 48
#define ORIG_SIZE 8

#define rightEnd (right+8)
#define bottomEnd (bottom+8)

__device__ void cuda_sad_block_8x8(uint8_t *block1, uint8_t *block2,
        int *result)
{
    int sum0 = 0; 
    int sum1 = 0; 
    int sum2 = 0; 
    int sum3 = 0; 
    int sum4 = 0; 
    int sum5 = 0; 
    int sum6 = 0; 
    int sum7 = 0;
    int minsad = INT_MAX;
    uint8_t *b1, *b20,*b21,*b22,*b23,*b24,*b25,*b26,*b27;

    b1 = block1;
    b20 = block2;
    b21 = block2 + 1;
    b22 = block2 + 2;
    b23 = block2 + 3;
    b24 = block2 + REF_WIDTH;
    b25 = block2 + REF_WIDTH + 1;
    b26 = block2 + REF_WIDTH + 2;
    b27 = block2 + REF_WIDTH + 3;
    int l1;
    //////////////////


    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    b20 += 40; b21 += 40; b22 += 40; b23 += 40; b24 += 40; b25 += 40; b26 += 40; b27 += 40;

    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    b20 += 40; b21 += 40; b22 += 40; b23 += 40; b24 += 40; b25 += 40; b26 += 40; b27 += 40;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    b20 += 40; b21 += 40; b22 += 40; b23 += 40; b24 += 40; b25 += 40; b26 += 40; b27 += 40;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    b20 += 40; b21 += 40; b22 += 40; b23 += 40; b24 += 40; b25 += 40; b26 += 40; b27 += 40;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    b20 += 40; b21 += 40; b22 += 40; b23 += 40; b24 += 40; b25 += 40; b26 += 40; b27 += 40;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    b20 += 40; b21 += 40; b22 += 40; b23 += 40; b24 += 40; b25 += 40; b26 += 40; b27 += 40;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    b20 += 40; b21 += 40; b22 += 40; b23 += 40; b24 += 40; b25 += 40; b26 += 40; b27 += 40;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    l1 = *b1; sum0 = __sad(l1, *b20, sum0); ++b20; sum1 = __sad(l1, *b21, sum1); ++b21; sum2 = __sad(l1, *b22, sum2); ++b22; sum3 = __sad(l1, *b23, sum3); ++b23; sum4 = __sad(l1, *b24, sum4); ++b24; sum5 = __sad(l1, *b25, sum5); ++b25; sum6 = __sad(l1, *b26, sum6); ++b26; sum7 = __sad(l1, *b27, sum7); ++b27; ++b1;
    b20 += 40; b21 += 40; b22 += 40; b23 += 40; b24 += 40; b25 += 40; b26 += 40; b27 += 40;


    ///////////////////
    minsad = min(minsad, (sum0 << 10));
    minsad = min(minsad, (sum1 << 10) + 1);
    minsad = min(minsad, (sum2 << 10) + 2);
    minsad = min(minsad, (sum3 << 10) + 3);
    minsad = min(minsad, (sum4 << 10) + 32);
    minsad = min(minsad, (sum5 << 10) + 32 + 1);
    minsad = min(minsad, (sum6 << 10) + 32 + 2);
    minsad = min(minsad, (sum7 << 10) + 32 + 3);

    // sadxy = sad*1024 + (mv_y+16)*32 + (mv_x+16)
    *result = minsad;
}

__global__ void k_me_block_8x8(uint8_t *orig, uint8_t *ref, int *mv_out, int w, int h)
{
    __shared__ int best_sadxy;
    __shared__ uint8_t shared_orig[ORIG_SIZE * ORIG_SIZE];
    __shared__ uint8_t shared_ref[REF_HEIGHT * REF_WIDTH];
    best_sadxy = INT_MAX;

    // copying ORIG global->shared
    int x = 4 * (threadIdx.x % 2) + (threadIdx.y / 2);
    int y = (threadIdx.x / 2) + 4 * (threadIdx.y % 2);
    if (threadIdx.y < 8) {
        shared_orig[y * ORIG_SIZE + x]
            = orig[(blockIdx.y * 8 + y) * w 
            + (blockIdx.x * 8 + x)];
    }

    int left = (blockIdx.x*8 - 16);
    int top = (blockIdx.y*8 - 16);
    int right = (blockIdx.x*8 + 16);
    int bottom = (blockIdx.y*8 + 16);

    // Make sure we are within bounds of reference frame
    // TODO: Support partial frame bounds
    if (left < 0)
        left = 0;
    if (top < 0)
        top = 0;
    if (right > (w - 8))
        right = w - 8;
    if (bottom > (h - 8))
        bottom = h - 8;

    //copying REF

    // 32 x 32
    x = 4 * threadIdx.x;
    y = 2 * threadIdx.y;
    for (int i = 0; i < 8; i++) {
        if (y < bottom - top && x < right - left) {
            int i_x = x + (i & 3);
            int i_y = y + (i >> 2);
            shared_ref[i_y * REF_WIDTH + i_x] = 
                ref[(top + i_y) * w + left + i_x];
        }
    }
    // bottom
    x = 4 * threadIdx.x + 2 * (threadIdx.y / 8);
    y = bottom - top + 1 * ((threadIdx.y % 8) / 4) 
        + 4 * ((threadIdx.y % 4) / 2) + 2 * (threadIdx.y % 2);
    
    if (x < right - left) {
        shared_ref[y * REF_WIDTH + x] = 
            ref[(top + y) * w + left + x];
        shared_ref[y * REF_WIDTH + x + 1] = 
            ref[(top + y) * w + left + x + 1];
    }
    
    // right
    x = 4 * (threadIdx.x % 2);
    y = 4 * (threadIdx.y / 2) + (threadIdx.x / 2);
    if (threadIdx.y % 2 == 0) {
        if (y < bottom - top) {
            for (int i = 0; i < 4; ++i) {
                int i_x = right - left + x + (i & 3);
                int i_y = y;
                shared_ref[i_y * REF_WIDTH + i_x]
                    = ref[(top + i_y) * w + left + i_x];
            }
        }
    }

    // corner
    x = right - left + 
        4 * (threadIdx.x % 2) + ((threadIdx.y / 2) / 2);
    y = bottom - top + 
        4 * ((threadIdx.y / 2) % 2) + (threadIdx.x / 2);
    if (threadIdx.y % 2 == 0) {
        shared_ref[y * REF_WIDTH + x]
            = ref[(top + y) * w + left + x];
    }
    
    __syncthreads();

    x = 4 * threadIdx.x;
    y = 2 * threadIdx.y;
    // SAD
    if (top + y < bottom && 
            left + x < right)
    {
        int mv_xy = (left + x - blockIdx.x * 8 + 16) 
            + ((top + y - blockIdx.y * 8 + 16) << 5);
        int sad;

        cuda_sad_block_8x8(shared_orig, shared_ref + y * REF_WIDTH + x, &sad);
        atomicMin(&best_sadxy, sad + mv_xy);
    }

    __syncthreads();

    // write out
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        mv_out[blockIdx.y * w / 8 + blockIdx.x] = best_sadxy;
    }
}

__global__ void k_mb(int *mv_out, struct macroblock *mbs,
        int padw, int padh)
{
    int mb_cols = padw / 8;
    int mb_rows = padh / 8;
    int mb_y = 16 * blockIdx.y + threadIdx.y;
    int mb_x = 8 * blockIdx.x + threadIdx.x;
    if (mb_x < mb_cols && mb_y < mb_rows) {
        int block_nr = mb_y * mb_cols + mb_x;
        // sadxy = sad*1024 + (mv_y+16)*32 + (mv_x+16)
        int sadxy = mv_out[block_nr];
        int sad = sadxy >> 10;
        int mv_y = ((sadxy >> 5) & 31) - 16;
        int mv_x = (sadxy & 31) - 16;
        struct macroblock *mb = &mbs[block_nr];
        if (sad < 512) {
            mb->use_mv = 1;
            mb->mv_x = mv_x;
            mb->mv_y = mv_y;
        }
        else {
            mb->use_mv = 0;
        }
    }
}

void cuda_me_cc(int padw, int padh, uint8_t *orig, uint8_t *ref,
        struct macroblock *mbs)
{
    /* Compare this frame with previous reconstructed frame */
    int *mv_out_dev;
    int mb_cols = padw / 8;
    int mb_rows = padh / 8;
    int blocks = mb_cols * mb_rows;
    hipMalloc(&mv_out_dev, blocks * sizeof(int));

    // Invoke kernel

    dim3 threadsPerBlock(8, 16);
    dim3 numBlocks(mb_cols, mb_rows);
    
    k_me_block_8x8<<<numBlocks, threadsPerBlock>>>
        (orig, ref, mv_out_dev, padw, padh);

    int blocks_width = (mb_cols + 7) / 8;
    int blocks_height = (mb_rows + 15) / 16;
    dim3 mb_blocks(blocks_width, blocks_height);

    k_mb<<<mb_blocks, threadsPerBlock>>> (mv_out_dev, mbs, padw, padh);

    hipFree(mv_out_dev);
}

void cuda_c63_motion_estimate(int width, int height,
        uint8_t *origY, uint8_t *origU, uint8_t *origV,
        uint8_t *reconsY, uint8_t *reconsU, uint8_t *reconsV,
        struct macroblock *mbsY, struct macroblock *mbsU,
        struct macroblock *mbsV) 
{
    //cudaPrintfInit();
    cuda_me_cc(width, height, origY, reconsY, mbsY);
    cuda_me_cc(width / 2, height / 2, origU, reconsU, mbsU);
    cuda_me_cc(width / 2, height / 2, origV, reconsV, mbsV);
    //cudaPrintfDisplay();
    //cudaPrintfEnd();
}

__global__ void k_mc_block_8x8(int w, int h, uint8_t *predicted, uint8_t *ref, struct macroblock *mbs) 
{
    int mb_x = blockIdx.x;
    int mb_y = blockIdx.y;
    struct macroblock *mb = &mbs[mb_y * w / 8 + mb_x];
    if (!mb->use_mv)
        return;

    /* Copy block from ref mandated by MV */
    int x = 8 * mb_x + threadIdx.x;
    int y = 8 * mb_y + threadIdx.y;
    predicted[y*w+x] = ref[(y + mb->mv_y) * w + (x + mb->mv_x)];
}

void cuda_c63_motion_compensate(int width, int height,
        uint8_t *reconsY, uint8_t *reconsU, uint8_t *reconsV,
        uint8_t *predY, uint8_t *predU, uint8_t *predV,
        struct macroblock *mbsY, struct macroblock *mbsU,
        struct macroblock *mbsV) 
{
    int mb_cols = width / 8;
    int mb_rows = height / 8;
    
    dim3 threads(8, 8);
    dim3 blocksY(mb_cols, mb_rows);
    dim3 blocksUV(mb_cols / 2, mb_rows / 2);

    k_mc_block_8x8<<<blocksY, threads>>> (width, height, 
            predY, reconsY, mbsY);
    k_mc_block_8x8<<<blocksUV, threads>>> (width / 2, height / 2, 
            predU, reconsU, mbsU);
    k_mc_block_8x8<<<blocksUV, threads>>> (width / 2, height / 2, 
            predV, reconsV, mbsV);
}


#include "c63.h"
#include "cuda_encode.h"
#include "cuda_me.h"
#include "cuda_dct.h"
#include "cuda_idct.h"
//#include "cuPrintf.cu"
    
void cuda_init_c63_encode(int width, int height,
        uint8_t **origY, uint8_t **origU, uint8_t **origV,
        uint8_t **reconsY, uint8_t **reconsU, uint8_t **reconsV,
        uint8_t **predY, uint8_t **predU, uint8_t **predV,
        int16_t **residY, int16_t **residU, int16_t **residV,
        struct macroblock **mbsY, struct macroblock **mbsU,
        struct macroblock **mbsV)
{
//    cudaPrintfInit();

    int ypw = 16 * ((width + 15) / 16);
    int yph = 16 * ((height + 15) / 16);
    int uvpw = 8 * ((width/2 + 7) / 8);
    int uvph = 8 * ((height/2 + 7) / 8);
    hipMalloc(origY, ypw * yph);
    hipMalloc(origU, uvpw * uvph);
    hipMalloc(origV, uvpw * uvph);
    hipMalloc(reconsY, ypw * yph);
    hipMalloc(reconsU, uvpw * uvph);
    hipMalloc(reconsV, uvpw * uvph);
    hipMalloc(predY, ypw * yph);
    hipMalloc(predU, uvpw * uvph);
    hipMalloc(predV, uvpw * uvph);
    hipMalloc(residY, ypw * yph * sizeof(int16_t));
    hipMalloc(residU, uvpw * uvph * sizeof(int16_t));
    hipMalloc(residV, uvpw * uvph * sizeof(int16_t));

    hipMalloc(mbsY, ypw * yph / 64 * sizeof(struct macroblock));
    hipMalloc(mbsU, uvpw * uvph / 64 * sizeof(struct macroblock));
    hipMalloc(mbsV, uvpw * uvph / 64 * sizeof(struct macroblock));
}


void cuda_free_c63_encode(
        uint8_t **origY, uint8_t **origU, uint8_t **origV,
        uint8_t **reconsY, uint8_t **reconsU, uint8_t **reconsV,
        uint8_t **predY, uint8_t **predU, uint8_t **predV,
        int16_t **residY, int16_t **residU, int16_t **residV,
        struct macroblock **mbsY, struct macroblock **mbsU,
        struct macroblock **mbsV)
{
    hipFree(*origY);
    hipFree(*origU);
    hipFree(*origV);
    hipFree(*reconsY);
    hipFree(*reconsU);
    hipFree(*reconsV);
    hipFree(*predY);
    hipFree(*predU);
    hipFree(*predV);
    hipFree(*residY);
    hipFree(*residU);
    hipFree(*residV);
    hipFree(*mbsY);
    hipFree(*mbsU);
    hipFree(*mbsV);

//    cudaPrintfDisplay();
//    cudaPrintfEnd();
}

void cuda_copy_image(int width, int height, yuv_t *image,
        uint8_t *origY, uint8_t *origU, uint8_t *origV)
{
    /*
    uint8_t t[7] = {0,1,2,3,4,5,6};
    uint8_t s[7];
    
    hipMalloc(&origY, 7);
    hipMemcpy(origY, t, 7, hipMemcpyHostToDevice);
    hipMemcpy(s, origY, 7, hipMemcpyDeviceToHost);
    printf("7: (%d, %d, %d, %d, %d, %d %d)\n", 
            s[0], s[1], s[2], s[3], s[4], s[5], s[6]);

*/

    hipMemcpy(origY, image->Y, width * height, 
            hipMemcpyHostToDevice);
    hipMemcpy(origU, image->U, width * height / 4, 
            hipMemcpyHostToDevice);
    hipMemcpy(origV, image->V, width * height / 4, 
            hipMemcpyHostToDevice);
   /* 
    uint8_t tab[352*288];
    int pos = 139 * width + 171;
    printf("(copy in: 171, 139-141) = %d, %d, %d\n",image->Y[pos],
            image->Y[pos+1], image->Y[pos+2]);

    hipMemcpy(tab, origY, width * height, 
            hipMemcpyHostToDevice);
    printf("(copy out: 171, 139-141) = %d, %d, %d\n",tab[pos],
            tab[pos+1], tab[pos+2]);
            */
}

void cuda_next_frame(int width, int height,
        uint8_t *origY, uint8_t *origU, uint8_t *origV,
        uint8_t *reconsY, uint8_t *reconsU, uint8_t *reconsV,
        uint8_t *predY, uint8_t *predU, uint8_t *predV,
        int16_t *residY, int16_t *residU, int16_t *residV,
        struct macroblock *mbsY, struct macroblock *mbsU,
        struct macroblock *mbsV)
{
    int ypw = 16 * ((width + 15) / 16);
    int yph = 16 * ((height + 15) / 16);
    int uvpw = 8 * ((width/2 + 7) / 8);
    int uvph = 8 * ((height/2 + 7) / 8);
    hipMemset(predY, 0x80, ypw * yph);
    hipMemset(predU, 0x80, uvpw * uvph);
    hipMemset(predV, 0x80, uvpw * uvph);
/*    hipMemset(residY, 0x80 ypw * yph * sizeof(int16_t));
    hipMemset(residU, 0x80, uvpw * uvph * sizeof(int16_t));
    hipMemset(residV, 0x80, uvpw * uvph * sizeof(int16_t));*/
    hipMemset(mbsY, 0, ypw * yph / 64
            * sizeof(struct macroblock));
    hipMemset(mbsU, 0, uvpw * uvph / 64 
            * sizeof(struct macroblock));
    hipMemset(mbsV, 0, uvpw * uvph / 64 
            * sizeof(struct macroblock));
}

void cuda_c63_encode_image(struct c63_common *cm, int width, int height, 
        uint8_t *origY, uint8_t *origU, uint8_t *origV,
        uint8_t *reconsY, uint8_t *reconsU, uint8_t *reconsV,
        uint8_t *predY, uint8_t *predU, uint8_t *predV,
        int16_t *residY, int16_t *residU, int16_t *residV,
        struct macroblock *mbsY, struct macroblock *mbsU,
        struct macroblock *mbsV)
{
    int ypw = 16 * ((width + 15) / 16);
    int yph = 16 * ((height + 15) / 16);
    int uvpw = 8 * ((width/2 + 7) / 8);
    int uvph = 8 * ((height/2 + 7) / 8);
/*    
    //DEBUG
    uint8_t tab[352*288];
    hipMemcpy(tab, reconsY, width * height, 
            hipMemcpyDeviceToHost);
    int pos = 139 * width + 171;
    printf("przed next Y: (171, 139-141) = %d, %d, %d\n",tab[pos],
            tab[pos+1], tab[pos+2]);
*/

    /* Advance to next frame */
    cuda_next_frame(width, height,
            origY, origU, origV, reconsY, reconsU, reconsV,
            predY, predU, predV, residY, residU, residV,
            mbsY, mbsU, mbsV);
  /*  
    //DEBUG
    hipMemcpy(tab, reconsY, width * height, 
            hipMemcpyDeviceToHost);
    printf("po next (Y: 171, 139-141) = %d, %d, %d\n",tab[pos],
            tab[pos+1], tab[pos+2]);
*/
    
       if (!cm->curframe->keyframe)
    {   
        cuda_c63_motion_estimate(ypw, yph,
                origY, origU, origV, reconsY, reconsU, reconsV,
                mbsY, mbsU, mbsV);
        cuda_c63_motion_compensate(ypw, yph,
                reconsY, reconsU, reconsV, predY, predU, predV,
                mbsY, mbsU, mbsV);
    }  
    
    cuda_dct_quantize(ypw, yph, 0, origY, predY, residY);
    cuda_dct_quantize(uvpw, uvph, 1, origU, predU, residU);
    cuda_dct_quantize(uvpw, uvph, 1, origV, predV, residV);

    cuda_dequantize_idct(ypw, yph, 0, residY, predY, reconsY);
    cuda_dequantize_idct(uvpw, uvph, 1, residU, predU, reconsU);
    cuda_dequantize_idct(uvpw, uvph, 1, residV, predV, reconsV);
    
    hipMemcpy(cm->curframe->residuals->Ydct, residY, 
            ypw * yph * sizeof(int16_t), hipMemcpyDeviceToHost);
    hipMemcpy(cm->curframe->residuals->Udct, residU, 
            uvpw * uvph * sizeof(int16_t), hipMemcpyDeviceToHost);
    hipMemcpy(cm->curframe->residuals->Vdct, residV, 
            uvpw * uvph * sizeof(int16_t), hipMemcpyDeviceToHost);
    
    hipMemcpy(cm->curframe->mbs[0], mbsY, ypw * yph / 64 *
            sizeof(struct macroblock), hipMemcpyDeviceToHost);
    hipMemcpy(cm->curframe->mbs[1], mbsU, uvpw * uvph / 64 *
            sizeof(struct macroblock), hipMemcpyDeviceToHost);
    hipMemcpy(cm->curframe->mbs[2], mbsV, uvpw * uvph / 64 *
            sizeof(struct macroblock), hipMemcpyDeviceToHost);
/*
    //DEBUG
    hipMemcpy(tab, origY, width * height, 
            hipMemcpyDeviceToHost);
    printf("(przed write Y: 171, 139-141) = %d, %d, %d\n",tab[pos],
            tab[pos+1], tab[pos+2]);
    //END DEBUG*/

    write_frame(cm);
    
}


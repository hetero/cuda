#include "c63.h"
#include "cuda_encode.h"
#include "cuda_me.h"
#include "cuda_dct.h"
    
void cuda_init_c63_encode(int width, int height,
        uint8_t *origY, uint8_t *origU, uint8_t *origV,
        uint8_t *reconsY, uint8_t *reconsU, uint8_t *reconsV,
        uint8_t *predY, uint8_t *predU, uint8_t *predV,
        int16_t *residY, int16_t *residU, int16_t *residV,
        struct macroblock *mbs[3])
{
    int ypw = 16 * ((width + 15) / 16);
    int yph = 16 * ((height + 15) / 16);
    int uvpw = 8 * ((width/2 + 7) / 8);
    int uvph = 8 * ((height/2 + 7) / 8);
    hipMalloc(&origY, ypw * yph);
    hipMalloc(&origU, uvpw * uvph);
    hipMalloc(&origV, uvpw * uvph);
    hipMalloc(&reconsY, ypw * yph);
    hipMalloc(&reconsU, uvpw * uvph);
    hipMalloc(&reconsV, uvpw * uvph);
    hipMalloc(&predY, ypw * yph);
    hipMalloc(&predU, uvpw * uvph);
    hipMalloc(&predV, uvpw * uvph);
    hipMalloc(&residY, ypw * yph * sizeof(int16_t));
    hipMalloc(&residU, uvpw * uvph * sizeof(int16_t));
    hipMalloc(&residV, uvpw * uvph * sizeof(int16_t));

    hipMalloc(&mbs[0], ypw * yph / 64 * sizeof(struct macroblock));
    hipMalloc(&mbs[1], uvpw * uvph / 64 * sizeof(struct macroblock));
    hipMalloc(&mbs[2], uvpw * uvph / 64 * sizeof(struct macroblock));
}


void cuda_free_c63_encode(
        uint8_t *origY, uint8_t *origU, uint8_t *origV,
        uint8_t *reconsY, uint8_t *reconsU, uint8_t *reconsV,
        uint8_t *predY, uint8_t *predU, uint8_t *predV,
        int16_t *residY, int16_t *residU, int16_t *residV,
        struct macroblock *mbs[3])
{
    hipFree(origY);
    hipFree(origU);
    hipFree(origV);
    hipFree(reconsY);
    hipFree(reconsU);
    hipFree(reconsV);
    hipFree(predY);
    hipFree(predU);
    hipFree(predV);
    hipFree(residY);
    hipFree(residU);
    hipFree(residV);
    hipFree(mbs[0]);
    hipFree(mbs[1]);
    hipFree(mbs[2]);
}

void cuda_copy_image(int width, int height, yuv_t *image,
        uint8_t *origY, uint8_t *origU, uint8_t *origV)
{
    hipMemcpy(origY, image->Y, width * height, 
            hipMemcpyHostToDevice);
    hipMemcpy(origU, image->U, width * height / 4, 
            hipMemcpyHostToDevice);
    hipMemcpy(origV, image->V, width * height / 4, 
            hipMemcpyHostToDevice);
}
void cuda_next_frame(int width, int height,
        uint8_t *origY, uint8_t *origU, uint8_t *origV,
        uint8_t *reconsY, uint8_t *reconsU, uint8_t *reconsV,
        uint8_t *predY, uint8_t *predU, uint8_t *predV,
        int16_t *residY, int16_t *residU, int16_t *residV,
        struct macroblock *mbs[3]) 
{
    int ypw = 16 * ((width + 15) / 16);
    int yph = 16 * ((height + 15) / 16);
    int uvpw = 8 * ((width/2 + 7) / 8);
    int uvph = 8 * ((height/2 + 7) / 8);
    hipMemset(predY, 0x80, ypw * yph);
    hipMemset(predU, 0x80, uvpw * uvph);
    hipMemset(predV, 0x80, uvpw * uvph);
    hipMemset(residY, 0x80, ypw * yph * sizeof(int16_t));
    hipMemset(residU, 0x80, uvpw * uvph * sizeof(int16_t));
    hipMemset(residV, 0x80, uvpw * uvph * sizeof(int16_t));
    hipMemset(mbs[0], 0, ypw * yph / 64
            * sizeof(struct macroblock));
    hipMemset(mbs[1], 0, uvpw * uvph / 64 
            * sizeof(struct macroblock));
    hipMemset(mbs[2], 0, uvpw * uvph / 64 
            * sizeof(struct macroblock));
}

void cuda_c63_encode_image(int keyframe, int width, int height, 
        uint8_t *origY, uint8_t *origU, uint8_t *origV,
        uint8_t *reconsY, uint8_t *reconsU, uint8_t *reconsV,
        uint8_t *predY, uint8_t *predU, uint8_t *predV,
        int16_t *residY, int16_t *residU, int16_t *residV,
        struct macroblock *mbs[3])
{
    int ypw = 16 * ((width + 15) / 16);
    int yph = 16 * ((height + 15) / 16);
    int uvpw = 8 * ((width/2 + 7) / 8);
    int uvph = 8 * ((height/2 + 7) / 8);
    /* Advance to next frame */
    cuda_next_frame(width, height,
            origY, origU, origV, reconsY, reconsU, reconsV,
            predY, predU, predV, residY, residU, residV,
            mbs); 

    if (!keyframe)
    {   
        cuda_c63_motion_estimate(ypw, yph,
            origY, origU, origV, reconsY, reconsU, reconsV,
            mbs);
        cuda_c63_motion_compensate(ypw, yph,
                reconsY, reconsU, reconsV, predY, predU, predV,
                mbs);
    }  
    
/*
    cuda_dct_quantize(image->Y, cm->curframe->predicted->Y, cm->padw[0], cm->padh[0], cm->curframe->residuals->Ydct, 0, dct_in_data_y, dct_prediction_y, dct_out_data_y);
    cuda_dct_quantize(image->U, cm->curframe->predicted->U, cm->padw[1], cm->padh[1], cm->curframe->residuals->Udct, 1, dct_in_data_uv, dct_prediction_uv, dct_out_data_uv);
    cuda_dct_quantize(image->V, cm->curframe->predicted->V, cm->padw[2], cm->padh[2], cm->curframe->residuals->Vdct, 1, dct_in_data_uv, dct_prediction_uv, dct_out_data_uv);

    cuda_dequantize_idct(cm->curframe->residuals->Ydct, cm->curframe->predicted->Y, cm->ypw, cm->yph, cm->curframe->recons->Y, 0, idct_in_data_y, idct_prediction_y, idct_out_data_y);
    cuda_dequantize_idct(cm->curframe->residuals->Udct, cm->curframe->predicted->U, cm->upw, cm->uph, cm->curframe->recons->U, 1, idct_in_data_uv, idct_prediction_uv, idct_out_data_uv);
    cuda_dequantize_idct(cm->curframe->residuals->Vdct, cm->curframe->predicted->V, cm->vpw, cm->vph, cm->curframe->recons->V, 1, idct_in_data_uv, idct_prediction_uv, idct_out_data_uv);
*/
    
    //write_frame(cm);

    //++cm->framenum;
    //++cm->frames_since_keyframe;
}


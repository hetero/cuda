#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <getopt.h>
#include <errno.h>
#include <stdint.h>
#include <math.h>
#include <assert.h>
#include <limits.h>

#include "c63.h"
#include "cuda_me.h"

__device__ void cuda_sad_block_8x8(uint8_t *block1, uint8_t *block2,
        int stride, int *result)
{
    *result = 0;
 
    int u,v;
    for (v=0; v<8; ++v)
        for (u=0; u<8; ++u)
            *result += abs(block2[v*stride+u] - block1[v*stride+u]);
}

__global__ void k_me_block_8x8(uint8_t *orig, uint8_t *ref, mv_out_t *mv_out, int w, int h)
{
    #define SAD_SIZE 32
    __shared__ int sad[SAD_SIZE * SAD_SIZE];

    int mb_x = blockIdx.x;
    int mb_y = blockIdx.y;
    int block_nr = mb_y * w / 8 + mb_x;
    
    int range = 16; //TODO

    int left = mb_x*8 - range;
    int top = mb_y*8 - range;
    int right = mb_x*8 + range;
    int bottom = mb_y*8 + range;

    // Make sure we are within bounds of reference frame
    // TODO: Support partial frame bounds
    if (left < 0)
        left = 0;
    if (top < 0)
        top = 0;
    if (right > (w - 8))
        right = w - 8;
    if (bottom > (h - 8))
        bottom = h - 8;


    int x = left + 2 * threadIdx.x;
    int y = top + threadIdx.y;

    //cuPrintf("(x,y) = (%d, %d)\n", x, y);
    int mx = mb_x * 8;
    int my = mb_y * 8;

    if (y<bottom && x<right)
    {
        cuda_sad_block_8x8(orig + my*w+mx, ref + y*w+x, w, 
                &sad[(y-top) * SAD_SIZE + (x-left)]);
        x++;
        cuda_sad_block_8x8(orig + my*w+mx, ref + y*w+x, w, 
                &sad[(y-top) * SAD_SIZE + (x-left)]);
    }

    __syncthreads();

    if (threadIdx.x == 0 && threadIdx.y == 0) {
        int best_sad = INT_MAX;
        int best_x, best_y;

        for (x = left; x < right; ++x) {
            for (y = top; y < bottom; ++y) {
//            printf("(%4d,%4d) - %d\n", x, y, sad);
                int sad_temp = sad[(y-top) * SAD_SIZE + (x-left)];
                if (sad_temp < best_sad)
                {
                    best_x = x - mx;
                    best_y = y - my;
                    best_sad = sad_temp;
                }
            }
        }
        mv_out[block_nr].sad = best_sad;
        mv_out[block_nr].mv_x = best_x;
        mv_out[block_nr].mv_y = best_y;
    }
}

void cuda_me_cc(struct c63_common *cm, int cc)
{
    /* Compare this frame with previous reconstructed frame */
    int mb_x, mb_y;

    uint8_t *orig, *ref;
    mv_out_t *mv_out_dev, *mv_out_host;
    int frame_size = cm->padw[cc] * cm->padh[cc];
    int mb_cols = cm->padw[cc] / 8;
    int mb_rows = cm->padh[cc] / 8;
    int blocks = mb_cols * mb_rows;
    hipMalloc(&orig, frame_size * sizeof(uint8_t));
    hipMalloc(&ref, frame_size * sizeof(uint8_t));
    hipMalloc(&mv_out_dev, blocks * sizeof(mv_out_t));

    mv_out_host = (mv_out_t *) malloc(blocks * sizeof(mv_out_t));

    // Copy vectors from host memory to device memory
    uint8_t *cur, *recons;
    switch (cc) {
        case 0:
            cur = cm->curframe->orig->Y;
            recons = cm->refframe->recons->Y;
            break;
        case 1:
            cur = cm->curframe->orig->U;
            recons = cm->refframe->recons->U;
            break;
        case 2:
            cur = cm->curframe->orig->V;
            recons = cm->refframe->recons->V;
    }
    
    hipMemcpy(orig, cur, frame_size, 
            hipMemcpyHostToDevice);
    hipMemcpy(ref, recons, frame_size, 
            hipMemcpyHostToDevice);
    
    // Invoke kernel

    dim3 threadsPerBlock(16, 32);
    dim3 numBlocks(mb_cols, mb_rows);

    k_me_block_8x8<<<numBlocks, threadsPerBlock>>>
        (orig, ref, mv_out_dev, cm->padw[cc], cm->padh[cc]); 
    
    // Copy result from device memory to host memory
    hipMemcpy(mv_out_host, mv_out_dev, blocks * sizeof(mv_out_t), 
            hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(orig);
    hipFree(ref);
    hipFree(mv_out_dev);

    for (mb_y = 0; mb_y < mb_rows; ++mb_y) {
        for (mb_x = 0; mb_x < mb_cols; ++mb_x) {
            int block_nr = mb_y * mb_cols + mb_x;
            int sad = mv_out_host[block_nr].sad;
            int mv_x = mv_out_host[block_nr].mv_x;
            int mv_y = mv_out_host[block_nr].mv_y;
            struct macroblock *mb = &cm->curframe->mbs[cc][block_nr];
            if (sad < 512) {
                mb->use_mv = 1;
                mb->mv_x = mv_x;
                mb->mv_y = mv_y;
            }
            else {
                mb->use_mv = 0;
            }
                //printf("(%d,%d): MV (%d, %d) with SAD %d\n", mb_x, mb_y, mb->mv_x, mb->mv_y, sad);
        }
    }
}

void cuda_c63_motion_estimate(struct c63_common *cm) {
    for (int cc = 0; cc <= 2; cc++) {
        cuda_me_cc(cm, cc);
    }
}

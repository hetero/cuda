#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <getopt.h>
#include <errno.h>
#include <stdint.h>
#include <math.h>
#include <assert.h>
#include <limits.h>
#include <sm_11_atomic_functions.h>
#include <sm_12_atomic_functions.h>
#include <sm_13_double_functions.h>

#include "c63.h"
#include "cuda_me.h"

#define REF_SIZE 39
#define ORIG_SIZE 8

__device__ void cuda_sad_block_8x8(uint8_t *block1, uint8_t *block2,
        int mv_xy, int *result)
{
    int res = 0;
    
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    block2 += 31;

    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    block2 += 31;

    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    block2 += 31;

    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    block2 += 31;

    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    block2 += 31;

    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    block2 += 31;

    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    block2 += 31;

    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    res = __sad(*block2, *block1, res); ++block1; ++block2;
    block2 += 31;

    // sadxy = sad*1024 + (mv_x+16)*32 + (mv_y+16)
    *result = (res << 10) + mv_xy;
}

__global__ void k_me_block_8x8(uint8_t *orig, uint8_t *ref, mv_out_t *mv_out, int w, int h)
{
    __shared__ int best_sadxy;
    __shared__ uint8_t shared_orig[ORIG_SIZE * ORIG_SIZE];
    __shared__ uint8_t shared_ref[REF_SIZE * REF_SIZE];
    best_sadxy = INT_MAX;

    int mb_x = blockIdx.x;
    int mb_y = blockIdx.y;
    int block_nr = mb_y * w / 8 + mb_x;
    
    int mx = mb_x * 8;
    int my = mb_y * 8;
    
    // copying ORIG global->shared
    if (threadIdx.x < ORIG_SIZE && threadIdx.y < ORIG_SIZE)
        shared_orig[threadIdx.y * ORIG_SIZE + threadIdx.x]
            = orig[(my+threadIdx.y) * w + (mx+threadIdx.x)];


    int range = 16; //TODO

    int left = mb_x*8 - range;
    int top = mb_y*8 - range;
    int right = mb_x*8 + range;
    int bottom = mb_y*8 + range;

    // Make sure we are within bounds of reference frame
    // TODO: Support partial frame bounds
    if (left < 0)
        left = 0;
    if (top < 0)
        top = 0;
    if (right > (w - 8))
        right = w - 8;
    if (bottom > (h - 8))
        bottom = h - 8;

    int rightEnd = right+7;
    int bottomEnd = bottom+7;

    //copying REF

    //1st whole block
    if (left + threadIdx.x < rightEnd && 
            top + threadIdx.y < bottomEnd) {
        shared_ref[threadIdx.y * REF_SIZE + threadIdx.x] =
            ref[(top + threadIdx.y) * w + (left + threadIdx.x)];
    }
    //2nd whole block
    if (left + 16 + threadIdx.x < rightEnd 
            && top + threadIdx.y < bottomEnd) {
        shared_ref[threadIdx.y * REF_SIZE + 16 + threadIdx.x] =
            ref[(top + threadIdx.y) * w + (left + 16 + threadIdx.x)];
    }

    //right border
    if (threadIdx.x < 7) {
        if (left + 32 + threadIdx.x < rightEnd
                && top + threadIdx.y < bottomEnd) {
            shared_ref[threadIdx.y * REF_SIZE + 32 + threadIdx.x] =
                ref[(top + threadIdx.y) * w + 
                    (left + 32 + threadIdx.x)];
        }
    }
    //bottom border
    else if (threadIdx.x < 14) {
        if (top + 32 + (threadIdx.x - 7) < bottomEnd 
                && left + threadIdx.y < rightEnd) {
            shared_ref[(32 + (threadIdx.x - 7)) * REF_SIZE +
                threadIdx.y] =
                ref[(top + 32 + (threadIdx.x - 7)) * w 
                    + (left + threadIdx.y)];
        }
    }
    //right-bottom corner
    else if ((threadIdx.y & 7) != 7) {
        int x = (threadIdx.y >> 3) + 4 * (threadIdx.x - 14);
        int y = threadIdx.y & 7;
        if (top + 32 + y < bottomEnd && left + 32 + x < rightEnd) {
            shared_ref[(32 + y) * REF_SIZE + (32 + x)] =
                ref[(top + 32 + y) * w + (left + 32 + x)];
        }
    }

    __syncthreads();

    int x = 2 * threadIdx.x;
    int y = threadIdx.y;

    if (top+y<bottom && left+x<right)
    {
        int sad1, sad2;
        int mv_xy = ((left+x-mx + 16) << 5) + (top+y-my + 16);
        cuda_sad_block_8x8(shared_orig, shared_ref + y*REF_SIZE+x, 
                mv_xy, &sad1);
        x++;
        cuda_sad_block_8x8(shared_orig, shared_ref + y*REF_SIZE+x, 
                mv_xy + 32, &sad2);
        atomicMin(&best_sadxy, min(sad1, sad2));
    }

    __syncthreads();

    mv_out[block_nr].sadxy = best_sadxy;
}

void cuda_me_cc(struct c63_common *cm, int cc)
{
    /* Compare this frame with previous reconstructed frame */
    int mb_x, mb_y;

    uint8_t *orig, *ref;
    mv_out_t *mv_out_dev, *mv_out_host;
    int frame_size = cm->padw[cc] * cm->padh[cc];
    int mb_cols = cm->padw[cc] / 8;
    int mb_rows = cm->padh[cc] / 8;
    int blocks = mb_cols * mb_rows;
    hipMalloc(&orig, frame_size * sizeof(uint8_t));
    hipMalloc(&ref, frame_size * sizeof(uint8_t));
    hipMalloc(&mv_out_dev, blocks * sizeof(mv_out_t));

    mv_out_host = (mv_out_t *) malloc(blocks * sizeof(mv_out_t));

    // Copy vectors from host memory to device memory
    uint8_t *cur, *recons;
    switch (cc) {
        case 0:
            cur = cm->curframe->orig->Y;
            recons = cm->refframe->recons->Y;
            break;
        case 1:
            cur = cm->curframe->orig->U;
            recons = cm->refframe->recons->U;
            break;
        case 2:
            cur = cm->curframe->orig->V;
            recons = cm->refframe->recons->V;
    }
    
    hipMemcpy(orig, cur, frame_size, 
            hipMemcpyHostToDevice);
    hipMemcpy(ref, recons, frame_size, 
            hipMemcpyHostToDevice);
    
    // Invoke kernel

    dim3 threadsPerBlock(16, 32);
    dim3 numBlocks(mb_cols, mb_rows);
    
    k_me_block_8x8<<<numBlocks, threadsPerBlock>>>
        (orig, ref, mv_out_dev, cm->padw[cc], cm->padh[cc]); 
    
    // Copy result from device memory to host memory
    hipMemcpy(mv_out_host, mv_out_dev, blocks * sizeof(mv_out_t), 
            hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(orig);
    hipFree(ref);
    hipFree(mv_out_dev);

    for (mb_y = 0; mb_y < mb_rows; ++mb_y) {
        for (mb_x = 0; mb_x < mb_cols; ++mb_x) {
            int block_nr = mb_y * mb_cols + mb_x;
            // sadxy = sad*1024 + (mv_x+16)*32 + (mv_y+16)
            int sadxy = mv_out_host[block_nr].sadxy;
            int sad = sadxy >> 10;
            int mv_x = ((sadxy >> 5) & 31) - 16;
            int mv_y = (sadxy & 31) - 16;
            struct macroblock *mb = &cm->curframe->mbs[cc][block_nr];
            //printf("(%d,%d): MV = (%d,%d), sad=%d\n",mb_x,mb_y,mv_x,
            //        mv_y,sad);
            if (sad < 512) {
                mb->use_mv = 1;
                mb->mv_x = mv_x;
                mb->mv_y = mv_y;
            }
            else {
                mb->use_mv = 0;
            }
        }
    }
}

void cuda_c63_motion_estimate(struct c63_common *cm) {
    for (int cc = 0; cc <= 2; cc++) {
        cuda_me_cc(cm, cc);
    }
}

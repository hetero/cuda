#include <list>
#include <pthread.h>

#include "c63.h"
#include "cuda_encode.h"
#include "cuda_me.h"
#include "cuda_dct.h"
#include "cuda_idct.h"
//#include "cuPrintf.cu"

using std::list;
extern list<pthread_t> th_id_list;
extern struct entropy_ctx write_entropy;
extern pthread_mutex_t mutex;
struct c63_common tmp_cm;

void *thread_write_frame(void *tmp_cm)
{
    struct c63_common *cm = (struct c63_common *)tmp_cm;
    cm->e_ctx = write_entropy;
    pthread_mutex_lock(&mutex);
    write_frame(cm);
    // small hack to remember entropy_ctx
    write_entropy = cm->e_ctx;
    destroy_cm_write(cm);
    pthread_mutex_unlock(&mutex);
    pthread_exit(NULL);
}


void cuda_init_c63_encode(int width, int height,
        uint8_t **origY, uint8_t **origU, uint8_t **origV,
        uint8_t **reconsY, uint8_t **reconsU, uint8_t **reconsV,
        uint8_t **predY, uint8_t **predU, uint8_t **predV,
        int16_t **residY, int16_t **residU, int16_t **residV,
        struct macroblock **mbsY, struct macroblock **mbsU,
        struct macroblock **mbsV)
{
//    cudaPrintfInit();

    int ypw = 16 * ((width + 15) / 16);
    int yph = 16 * ((height + 15) / 16);
    int uvpw = 8 * ((width/2 + 7) / 8);
    int uvph = 8 * ((height/2 + 7) / 8);
    hipMalloc(origY, ypw * yph);
    hipMalloc(origU, uvpw * uvph);
    hipMalloc(origV, uvpw * uvph);
    hipMalloc(reconsY, ypw * yph);
    hipMalloc(reconsU, uvpw * uvph);
    hipMalloc(reconsV, uvpw * uvph);
    hipMalloc(predY, ypw * yph);
    hipMalloc(predU, uvpw * uvph);
    hipMalloc(predV, uvpw * uvph);
    hipMalloc(residY, ypw * yph * sizeof(int16_t));
    hipMalloc(residU, uvpw * uvph * sizeof(int16_t));
    hipMalloc(residV, uvpw * uvph * sizeof(int16_t));

    hipMalloc(mbsY, ypw * yph / 64 * sizeof(struct macroblock));
    hipMalloc(mbsU, uvpw * uvph / 64 * sizeof(struct macroblock));
    hipMalloc(mbsV, uvpw * uvph / 64 * sizeof(struct macroblock));
}


void cuda_free_c63_encode(
        uint8_t **origY, uint8_t **origU, uint8_t **origV,
        uint8_t **reconsY, uint8_t **reconsU, uint8_t **reconsV,
        uint8_t **predY, uint8_t **predU, uint8_t **predV,
        int16_t **residY, int16_t **residU, int16_t **residV,
        struct macroblock **mbsY, struct macroblock **mbsU,
        struct macroblock **mbsV)
{
    hipFree(*origY);
    hipFree(*origU);
    hipFree(*origV);
    hipFree(*reconsY);
    hipFree(*reconsU);
    hipFree(*reconsV);
    hipFree(*predY);
    hipFree(*predU);
    hipFree(*predV);
    hipFree(*residY);
    hipFree(*residU);
    hipFree(*residV);
    hipFree(*mbsY);
    hipFree(*mbsU);
    hipFree(*mbsV);

//    cudaPrintfDisplay();
//    cudaPrintfEnd();
}

void cuda_copy_image(int width, int height, yuv_t *image,
        uint8_t *origY, uint8_t *origU, uint8_t *origV)
{
    hipMemcpy(origY, image->Y, width * height, 
            hipMemcpyHostToDevice);
    hipMemcpy(origU, image->U, width * height / 4, 
            hipMemcpyHostToDevice);
    hipMemcpy(origV, image->V, width * height / 4, 
            hipMemcpyHostToDevice);
}

void cuda_next_frame(int width, int height,
        uint8_t *origY, uint8_t *origU, uint8_t *origV,
        uint8_t *reconsY, uint8_t *reconsU, uint8_t *reconsV,
        uint8_t *predY, uint8_t *predU, uint8_t *predV,
        int16_t *residY, int16_t *residU, int16_t *residV,
        struct macroblock *mbsY, struct macroblock *mbsU,
        struct macroblock *mbsV)
{
    int ypw = 16 * ((width + 15) / 16);
    int yph = 16 * ((height + 15) / 16);
    int uvpw = 8 * ((width/2 + 7) / 8);
    int uvph = 8 * ((height/2 + 7) / 8);
    hipMemset(predY, 0x80, ypw * yph);
    hipMemset(predU, 0x80, uvpw * uvph);
    hipMemset(predV, 0x80, uvpw * uvph);
    hipMemset(mbsY, 0, ypw * yph / 64
            * sizeof(struct macroblock));
    hipMemset(mbsU, 0, uvpw * uvph / 64 
            * sizeof(struct macroblock));
    hipMemset(mbsV, 0, uvpw * uvph / 64 
            * sizeof(struct macroblock));
}

void cuda_c63_encode_image(struct c63_common *cm, int width, int height, 
        uint8_t *origY, uint8_t *origU, uint8_t *origV,
        uint8_t *reconsY, uint8_t *reconsU, uint8_t *reconsV,
        uint8_t *predY, uint8_t *predU, uint8_t *predV,
        int16_t *residY, int16_t *residU, int16_t *residV,
        struct macroblock *mbsY, struct macroblock *mbsU,
        struct macroblock *mbsV)
{
    int ypw = 16 * ((width + 15) / 16);
    int yph = 16 * ((height + 15) / 16);
    int uvpw = 8 * ((width/2 + 7) / 8);
    int uvph = 8 * ((height/2 + 7) / 8);

    /* Advance to next frame */
    cuda_next_frame(width, height,
            origY, origU, origV, reconsY, reconsU, reconsV,
            predY, predU, predV, residY, residU, residV,
            mbsY, mbsU, mbsV);
    
       if (!cm->curframe->keyframe)
    {   
        cuda_c63_motion_estimate(ypw, yph,
                origY, origU, origV, reconsY, reconsU, reconsV,
                mbsY, mbsU, mbsV);
        cuda_c63_motion_compensate(ypw, yph,
                reconsY, reconsU, reconsV, predY, predU, predV,
                mbsY, mbsU, mbsV);
    }  
    
    cuda_dct_quantize(ypw, yph, 0, origY, predY, residY);
    cuda_dct_quantize(uvpw, uvph, 1, origU, predU, residU);
    cuda_dct_quantize(uvpw, uvph, 1, origV, predV, residV);

    cuda_dequantize_idct(ypw, yph, 0, residY, predY, reconsY);
    cuda_dequantize_idct(uvpw, uvph, 1, residU, predU, reconsU);
    cuda_dequantize_idct(uvpw, uvph, 1, residV, predV, reconsV);
    
    hipMemcpy(cm->curframe->residuals->Ydct, residY, 
            ypw * yph * sizeof(int16_t), hipMemcpyDeviceToHost);
    hipMemcpy(cm->curframe->residuals->Udct, residU, 
            uvpw * uvph * sizeof(int16_t), hipMemcpyDeviceToHost);
    hipMemcpy(cm->curframe->residuals->Vdct, residV, 
            uvpw * uvph * sizeof(int16_t), hipMemcpyDeviceToHost);
    
    hipMemcpy(cm->curframe->mbs[0], mbsY, ypw * yph / 64 *
            sizeof(struct macroblock), hipMemcpyDeviceToHost);
    hipMemcpy(cm->curframe->mbs[1], mbsU, uvpw * uvph / 64 *
            sizeof(struct macroblock), hipMemcpyDeviceToHost);
    hipMemcpy(cm->curframe->mbs[2], mbsV, uvpw * uvph / 64 *
            sizeof(struct macroblock), hipMemcpyDeviceToHost);
    
    pthread_t t;
    tmp_cm = *cm;
    pthread_create(&t, NULL, thread_write_frame, (void*)&tmp_cm);
    th_id_list.push_back(t);
    cuda_fake_cm_init(cm); 
    
    //write_frame(cm);
}


